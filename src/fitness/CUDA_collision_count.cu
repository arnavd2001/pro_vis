#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

extern "C" {
	#include "CUDA_header.h"
}

/* Divides 'dividend' by 'divisor', rounding up.
 */
static inline
int divisionCeil(int dividend, int divisor){
	return (dividend + divisor - 1) / divisor;
}

// Returns the first power of 2 that is >= 'base'.
static inline
int higherEqualPow2(int base){
	int result = 1;
	while(result < base) result <<= 1;
	return result;
}

/* Multi-block reduce.
 * Accepts only vectors that are power of 2.
 */
__global__ static
void reduce(int *vec, int *result){
	extern __shared__ int sdata[];

	int tid = threadIdx.x;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	sdata[tid] = vec[idx];
	__syncthreads();

	// Reduce
	for(int stride = blockDim.x >> 1; stride > 0; stride >>= 1){
		if(threadIdx.x < stride)
			sdata[threadIdx.x] += sdata[threadIdx.x+stride];

		__syncthreads();
	}

	result[blockIdx.x] = sdata[0];
}


/*
 * Collision Count procedure implemented in CUDA.
 *
 * This procedure parallelizes the sequential algorithm:
 * for i in 0:N-2
 *   for j in i+1:N-1
 *     collisions += (bead[i] == bead[j])
 * by performing just the outer 'for' in parallel.
 */
__global__
void count_collisions_cu(float3 *coords, int *result, int nCoords, int star){
	int baseIdx = blockIdx.x * 1024;
	int horizontalId = threadIdx.x + blockIdx.x * blockDim.x;

	// We read our element in a register (surplus threads will read anything)
	float3 buf = coords[horizontalId % nCoords];

	// Read first 2 blocks into shared memory
	extern __shared__ float3 sCoords[];
	sCoords[threadIdx.x] = coords[ (baseIdx + threadIdx.x) % nCoords ];
	sCoords[threadIdx.x + 1024] = coords[ (baseIdx + threadIdx.x + 1024) % nCoords ];
	__syncthreads();
	
	// Move our base index
	baseIdx = baseIdx + 2048; // We could use modulus here, but doesn't seem necessary

	// Count collisions
	int iterations = 0;
	int collisions = 0;
	int offset = 1;
	while(iterations < star){
		// Do 1024 iterations, or maybe less
		int limit = min(iterations + 1024, star);
		for(; iterations < limit; iterations++){
			// We want to check if sqrt( (Vx - Vy)(Vx - Vy) ) <= 1    Vx and Vy are float3 vectors
			// Which is the same as      (Vx - Vy)(Vx - Vy)   <= 1    and the product is an inner product
			// So we begin by taking the difference
			float3 diff = make_float3(
					buf.x - sCoords[threadIdx.x + offset].x,
					buf.y - sCoords[threadIdx.x + offset].y,
					buf.z - sCoords[threadIdx.x + offset].z
				);

			// horizontalId + iterations + 1 is the element we are comparing to
			if(diff.x*diff.x + diff.y*diff.y + diff.z*diff.z <= 1){
				collisions += 1;
			}

			offset++;
		}
		
		// If offset == 1025, this means beads in shared memory need to be replaced
		if(offset == 1025){
			// Change blocks in shared memory when needed
			// Unfortunately we need to synchronize threads here
			__syncthreads();
			
			// Rewrite older block with earlier block
			sCoords[threadIdx.x] = sCoords[threadIdx.x + 1024];
			// Read new block
			sCoords[threadIdx.x + 1024] = coords[ (baseIdx + threadIdx.x) % nCoords ];

			// We also have to sync here
			__syncthreads();
			
			// Move base index
			baseIdx += 1024;

			offset = 1;
		}
	}

	// If the vector has an even number of elements
	// Because of this, half of the elements must execute one more iteration
	// Notice that the way the 'for...loop' above was implemented, when the
	//   code reach this point, the shared memory has valid elements for one
	//   more iteration, so we don't need to verify it again.
	// Do one more iteration:
	if(horizontalId < nCoords/2 && nCoords%2 == 0){
		float3 diff = make_float3(
				buf.x - sCoords[threadIdx.x + offset].x,
				buf.y - sCoords[threadIdx.x + offset].y,
				buf.z - sCoords[threadIdx.x + offset].z
			);

		// horizontalId + iterations + 1 is the element we are comparing to
		if(diff.x*diff.x + diff.y*diff.y + diff.z*diff.z <= 1){
			collisions += 1;
		}
		offset++;
		iterations++;
	}

	// Sync before reducing collisions on shared memory
	__syncthreads();

	// Fill shared memory with collisions
	// We ignore collision from surplus threads
	extern __shared__ int sdata[];
	sdata[threadIdx.x] = collisions * (horizontalId < nCoords);
	__syncthreads();

	// Reduce 1024 elements
	for(int stride = 512; stride > 0; stride >>= 1){
		if(threadIdx.x < stride)
			sdata[threadIdx.x] += sdata[threadIdx.x+stride];

		__syncthreads();
	}

	// Export result
	if(threadIdx.x == 0){
		result[blockIdx.x] = sdata[0];
	}
}

/* Gets the next cuda stream in the circular list of streams.
 */
static
hipStream_t get_next_stream(){
	const int nStreams = 8;
	static hipStream_t streams[nStreams];
	static unsigned int launches = 0;

	// Allocate cuda streams in the first execution
	static int streamInit = 0;
	if(streamInit == 0){
		streamInit = 1;
		for(int i = 0; i < nStreams; i++){
			hipStreamCreate(&streams[i]);
		}
	}

	launches++;
	return streams[launches%nStreams];
}

/* Given a vector with 3D coordinates of points in the space,
 *   this function calculates the number of collisions among
 *   points, using CUDA-enable GPU.
 *
 * This functions just launches the kernel, returning a
 *   structure that can later be used to fetch the result
 *   back from the device memory.
 */
extern "C" struct CollisionCountPromise
count_collisions_launch(ElfFloat3d *vector, int size){
	if(size == 0){
		CollisionCountPromise retval = { NULL, NULL };
		return retval;
	}

	float3 *d_vector;
	int *d_result;
	hipStream_t stream = get_next_stream();

	// Allocate cuda vector for the 3D coordinates
	hipMalloc(&d_vector, sizeof(float3) * size);
	hipMemcpyAsync(d_vector, vector, sizeof(float3) * size, hipMemcpyHostToDevice, stream);

	// Prepare kernel launch parameters
	const int elemInShmem = 2048; // 2048 because we need 2 blocks of 1024 elements in shmem.
	int nThreads = 1024;          // We allocate maximum number of threads per block.
	int nBlocks = divisionCeil(size, nThreads);
	int nShMem = elemInShmem * sizeof(float3); // Shared memory required

	// Calculate the number of iterations S* (S star)
	// It is the number of iterations where --all-- threads execute work
	int star;
	if(size%2 == 0)
		star = (size - 2)/2;
	else star = (size - 1)/2;

	// Allocate cuda memory for the number of collisions
	// This will also be used as a working vector for reducing among blocks
	int resultSize = higherEqualPow2(nBlocks);
	hipMalloc(&d_result, sizeof(int) * resultSize);
	hipMemsetAsync(d_result, 0, sizeof(int) * resultSize, stream); // Reset is needed due to size overestimation

	// Finally launch kernels
	count_collisions_cu<<<nBlocks, nThreads, nShMem, stream>>>(d_vector, d_result, size, star);

	// Reduce the result vector
	nBlocks = resultSize/1024;
	int workSize = resultSize;
	int *d_toReduce = d_result;
	int *d_reduced  = (int *) d_vector;
	while(true){
		if(nBlocks == 0){
			reduce<<<1, workSize, sizeof(int) * workSize, stream>>>(d_toReduce, d_reduced);
			break;
		}

		reduce<<<nBlocks, 1024, sizeof(int) * 1024, stream>>>(d_toReduce, d_reduced);

		// For the next run, vectors should be swapped
		int *aux = d_reduced;
		d_reduced = d_toReduce;
		d_toReduce = aux;

		// For the next run, the workSize and nBlocks are lower
		workSize = nBlocks;
		nBlocks = workSize/1024;
	}

	const struct CollisionCountPromise ret = { d_toReduce, d_reduced };
	return ret;
}

/* This procedure fetches the result from the call to the
 *   _launch correspondent.
 * The pointers within the promise structure are freed, so
 *   it shouldn't be used anywhere after a call to this function.
 */
extern "C" int count_collisions_fetch(struct CollisionCountPromise promise){
	if(promise.d_toReduce == NULL && promise.d_reduced == NULL)
		return 0;

	const int n = 1;
	int result[n];
	hipMemcpy(&result, promise.d_reduced, sizeof(int) * n, hipMemcpyDeviceToHost);

	hipFree(promise.d_toReduce);
	hipFree(promise.d_reduced);

	return result[0];
}
